#include "hip/hip_runtime.h"
#include "ctranslate2/ops/conv2d.h"
#include "ctranslate2/ops/gemm.h"

#include "type_dispatch.h"
#include "cuda/helpers.h"
#include "cuda/utils.h"

// #include <iostream>

namespace ctranslate2 {
  namespace ops {

    template <typename T>
    __global__ void add_bias_kernel(const dim_t size, const dim_t channels, const dim_t output_height, const dim_t output_width, const T* bias, T* output) {
      const dim_t index = blockIdx.x * blockDim.x + threadIdx.x;
      if (index < size) {
        output[index] += bias[(index / (output_height * output_width)) % channels];
      }
    }

    template <Device D, typename T>
    void Conv2D::compute(const StorageView& input,
                         const StorageView& weight,
                         const StorageView* bias,
                         StorageView& output,
                         const StorageView* qscale) const {
      if (qscale) {
        throw std::runtime_error("Quantization is not supported in this Conv2D implementation");
      }

      // Input dimensions: (batch_size, in_channels, input_height, input_width)
      const dim_t batch_size = input.dim(0);
      const dim_t in_channels = input.dim(1);
      const dim_t input_height = input.dim(2);
      const dim_t input_width = input.dim(3);

      // Weight dimensions: (out_channels, in_channels, kernel_height, kernel_width)
      const dim_t out_channels = weight.dim(0);
      const dim_t kernel_height = weight.dim(2);
      const dim_t kernel_width = weight.dim(3);

      // Calculate output dimensions
      const dim_t output_height = (input_height + 2 * _padding_h - (_dilation * (kernel_height - 1) + 1)) / _stride_h + 1;
      const dim_t output_width = (input_width + 2 * _padding_w - (_dilation * (kernel_width - 1) + 1)) / _stride_w + 1;

      // Resize output tensor
      output.resize({batch_size, out_channels, output_height, output_width});

      StorageView im2col_output({batch_size, output_height * output_width, in_channels * kernel_height * kernel_width}, 0.0f, weight.device());
      im2col_transposed_gpu(input, im2col_output, kernel_height, kernel_width);
      // std::cout << "im2col_output: " << im2col_output.to(Device::CPU) << ", " << output_height << ", " << output_width << std::endl;

      StorageView weight_view(weight.dtype(), weight.device());
      weight_view.view(const_cast<void*>(weight.buffer()), {weight.dim(0), in_channels * kernel_height * kernel_width});
      // std::cout << "weight.dtype: " << (int)(weight.dtype()) << " weight.device: " << (int)(weight.device()) << std::endl;
      // std::cout << "weight_view: " << weight_view.to(Device::CPU) << ", " << output_height << ", " << output_width << std::endl;

      const dim_t m = out_channels;
      const dim_t n = output_height * output_width;
      const dim_t k = in_channels * kernel_height * kernel_width;
      const dim_t strideb = k * output_height * output_width;
      const dim_t stridec = out_channels * output_height * output_width;
      auto* b = im2col_output.data<float>();
      auto* c = output.data<float>();

      const Gemm gemm(1.0, 0.0, false, true);
      const auto device = im2col_output.device();
      for (dim_t i = 0; i < batch_size; ++i) {
        float* b_i = b + (i * strideb);
        float* c_i = c + (i * stridec);
        StorageView bb({n, k}, b_i, device); // transposed
        StorageView cc({m, n}, c_i, device);

        // std::cout << "bb: " << bb.to(Device::CPU) << std::endl;
        // std::cout << "cc: " << cc.to(Device::CPU) << std::endl;
        gemm(weight_view, bb, cc);
        // std::cout << "cc: " << cc.to(Device::CPU) << std::endl;

      }
      // std::cout << "gemm: " << output.to(Device::CPU) << std::endl;

      // Add bias
      if (bias) {
        const float* bias_data = bias->data<float>();
        float* output_data = output.data<float>();

        const dim_t size = output.size();
        const dim_t channels = bias->size();
        // std::cout << "size: " << size << " channels: " << channels << " out_channels: " << out_channels << std::endl;

        const dim3 block_dim(256);
        const dim3 grid_dim((size + block_dim.x - 1) / block_dim.x);

        add_bias_kernel<<<grid_dim, block_dim, 0, cuda::get_cuda_stream()>>>(
          size,
          out_channels,
          output_height,
          output_width,
          cuda::device_cast(bias_data),
          cuda::device_cast(output_data)
        );
        hipDeviceSynchronize();
        // std::cout << "bias: " << bias->to(Device::CPU) << std::endl;
        // std::cout << "output: " << output.to(Device::CPU) << std::endl;
      }
    }

    // im2col kernel adapted from candle:
    // https://github.com/huggingface/candle/blob/6f0b807ffd553fed27325a2a118b0e30bb6d9cbd/candle-kernels/src/conv.cu
    //  Apache-2.0 OR MIT license
    template <typename T>
    __device__ void im2col(
        const size_t dst_numel,
        const size_t h_out,
        const size_t w_out,
        const size_t h_k,
        const size_t w_k,
        const size_t stride_h,
        const size_t stride_w,
        const size_t padding_h,
        const size_t padding_w,
        const size_t dilation,
        const int64_t b_size,
        const int64_t c_in,
        const int64_t h_in,
        const int64_t w_in,
        const T *src,
        T *dst
    ) {
        const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
        // dst: (b_size, h_out, w_out, c_in, h_k, w_k)
        // src: (b_size, c_in, h_in, w_in)
        if (dst_i >= dst_numel) {
            return;
        }
        const size_t src_s0 = c_in * h_in * w_in;
        const size_t src_s1 = h_in * w_in;
        const size_t src_s2 = w_in;
        const size_t src_s3 = 1;

        const size_t dst_s4 = w_k;
        const size_t dst_s3 = h_k * dst_s4;
        const size_t dst_s2 = c_in * dst_s3;
        const size_t dst_s1 = w_out * dst_s2;
        const size_t dst_s0 = h_out * dst_s1;

        size_t tmp_dst_i = dst_i;
        const size_t b_idx = tmp_dst_i / dst_s0;
        tmp_dst_i -= b_idx * dst_s0;
        const size_t h_idx = tmp_dst_i / dst_s1;
        tmp_dst_i -= h_idx * dst_s1;
        const size_t w_idx = tmp_dst_i / dst_s2;
        tmp_dst_i -= w_idx * dst_s2;
        const size_t c_idx = tmp_dst_i / dst_s3;
        tmp_dst_i -= c_idx * dst_s3;
        const size_t h_k_idx = tmp_dst_i / dst_s4;
        tmp_dst_i -= h_k_idx * dst_s4;
        const size_t w_k_idx = tmp_dst_i;

        size_t src_h_idx = h_idx * stride_h + h_k_idx * dilation;
        size_t src_w_idx = w_idx * stride_w + w_k_idx * dilation;
        
        if (src_h_idx < padding_h || src_h_idx >= h_in + padding_h) {
            dst[dst_i] = static_cast<T>(0);
        }
        else if (src_w_idx < padding_w || src_w_idx >= w_in + padding_w) {
            dst[dst_i] = static_cast<T>(0);
        }
        else {
            src_h_idx -= padding_h;
            src_w_idx -= padding_w;
            const size_t src_i =
            b_idx * src_s0
            + c_idx * src_s1
            + src_h_idx * src_s2
            + src_w_idx * src_s3;
            dst[dst_i] = src[src_i];
        }
    }

    template <typename T>
    __global__ void im2col_kernel(
        const size_t dst_numel,
        const size_t h_out,
        const size_t w_out,
        const size_t h_k,
        const size_t w_k,
        const size_t stride_h,
        const size_t stride_w,
        const size_t padding_h,
        const size_t padding_w,
        const size_t dilation,
        const int64_t info0,
        const int64_t info1,
        const int64_t info2,
        const int64_t info3,
        const T *src,
        T *dst) {
        im2col<T>(
            dst_numel,
            h_out,
            w_out,
            h_k,
            w_k,
            stride_h,
            stride_w,
            padding_h,
            padding_w,
            dilation,
            info0,
            info1,
            info2,
            info3,
            src,
            dst);
    }

    void Conv2D::im2col_transposed_gpu(const StorageView& input, StorageView& output, const dim_t kernel_height, const dim_t kernel_width) const {
      const size_t dst_numel = output.size();
      const size_t h_out = (input.dim(2) + 2 * _padding_h - (_dilation * (kernel_height - 1) + 1)) / _stride_h + 1;
      const size_t w_out = (input.dim(3) + 2 * _padding_w - (_dilation * (kernel_width - 1) + 1)) / _stride_w + 1;
      const int64_t *info = input.shape().data();
      const float* src = input.data<float>();
      float* dst = output.data<float>();

      const dim3 block_dim(256);
      const dim3 grid_dim((dst_numel + block_dim.x - 1) / block_dim.x);

      im2col_kernel<<<grid_dim, block_dim, 0, cuda::get_cuda_stream()>>>(
          dst_numel,
          h_out,
          w_out,
          kernel_height,
          kernel_width,
          _stride_h,
          _stride_w,
          _padding_h,
          _padding_w,
          _dilation,
          info[0],
          info[1],
          info[2],
          info[3],
          src,
          dst);
      hipError_t err = hipGetLastError();
      if (err != hipSuccess) {
          std::cerr << "CUDA error after kernel launch: " << hipGetErrorString(err) << std::endl;
          abort();
      }

      hipDeviceSynchronize();
      err = hipGetLastError();
      if (err != hipSuccess) {
          std::cerr << "CUDA error after device synchronization: " << hipGetErrorString(err) << std::endl;
          abort();
      }
    }

#define DECLARE_IMPL(T)                                                 \
    template void                                                       \
    Conv2D::compute<Device::CUDA, T>(const StorageView& input,          \
                                     const StorageView& weight,         \
                                     const StorageView* bias,           \
                                     StorageView& output,               \
                                     const StorageView* qscale) const;

    DECLARE_IMPL(float)
    DECLARE_IMPL(float16_t)
    DECLARE_IMPL(bfloat16_t)

  }
}
